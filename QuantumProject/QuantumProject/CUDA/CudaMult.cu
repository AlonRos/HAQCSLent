#include "hip/hip_runtime.h"
#include "../CUDA/CudaHeader.cuh"
#include <chrono>


typedef struct {
	int width;
	int height;
	int jump;
	hipDoubleComplex* elements;

} GpuMatrix;

__device__ 
hipDoubleComplex getElement(GpuMatrix& A, int row, int col) {
	return A.elements[row * A.jump + col];
}
__device__ 
void setElement(GpuMatrix& A, int row, int col, hipDoubleComplex value) {
	A.elements[row * A.jump + col] = value;
}

__device__ 
GpuMatrix getSubMatrix(GpuMatrix& A, int row, int col, int blockHeightA, int blockWidthA)
{
	GpuMatrix Asub;
	Asub.width = blockWidthA;
	Asub.height = blockHeightA;
	Asub.jump = A.jump;
	Asub.elements = &A.elements[A.jump * blockHeightA * row + blockWidthA * col]; // staritng in the element (row, col)
	return Asub;
}

__global__ 
void matMulKernel(GpuMatrix A, GpuMatrix B, GpuMatrix C, int blockHeightA, int blockWidthAHeightB, int blockWidthB) {
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	GpuMatrix subC = getSubMatrix(C, blockRow, blockCol, blockHeightA, blockWidthB);

	hipDoubleComplex Cvalue = make_hipDoubleComplex(0, 0);

	int row = threadIdx.y;
	int col = threadIdx.x;

	__shared__ hipDoubleComplex As[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE]; // shared sub matrix of A
	__shared__ hipDoubleComplex Bs[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE]; // shared sub matrix of B

	for (int i = 0; i < A.width / blockWidthAHeightB; ++i) {
		GpuMatrix subA = getSubMatrix(A, blockRow, i, blockHeightA, blockWidthAHeightB); // get the current sub matrix of A

		GpuMatrix subB = getSubMatrix(B, i, blockCol, blockWidthAHeightB, blockWidthB); // get the current sub matrix of B

		if (row < subA.height && col < subA.width) {
			As[row][col] = getElement(subA, row, col); // copy from subA to the shared memory
		}

		if (row < subB.height && col < subB.width) {
			Bs[row][col] = getElement(subB, row, col); // copy from subB to the shared memory
		}

		__syncthreads(); // wait for all threads

		if (row < subC.height && col < subC.width) {
			for (int j = 0; j < blockWidthAHeightB; ++j) { // compute the row times the col
				Cvalue = hipCadd(Cvalue, hipCmul(As[row][j], Bs[j][col]));
			}
		}

		__syncthreads(); // wait for all threads
	}

	if (row < subC.height && col < subC.width) {
		setElement(subC, row, col, Cvalue); // save the result
	}
}

__host__
hipDoubleComplex* gpuMultArrs(hipDoubleComplex* A, int Am, int An, hipDoubleComplex* B, int Bn) {
	GpuMatrix dev_A, dev_B, dev_res;
	int Bm = An, resm = Am, resn = Bn;

	// copy hipDoubleComplex* A to GpuMatrix dev_A
	dev_A.width = dev_A.jump = An;
	dev_A.height = Am;
	size_t size = dev_A.width * dev_A.height * sizeof(hipDoubleComplex);
	hipMalloc(&dev_A.elements, size);
	hipMemcpy(dev_A.elements, A, size, hipMemcpyHostToDevice);

	// copy hipDoubleComplex* B to GpuMatrix dev_B
	dev_B.width = dev_B.jump = Bn;
	dev_B.height = Bm;
	size = dev_B.width * dev_B.height * sizeof(hipDoubleComplex);
	hipMalloc(&dev_B.elements, size);
	hipMemcpy(dev_B.elements, B, size, hipMemcpyHostToDevice);


	// alloc gpu memory for the result
	dev_res.width = dev_res.jump = resn;
	dev_res.height = resm;
	size = dev_res.width * dev_res.height * sizeof(hipDoubleComplex);
	hipMalloc(&dev_res.elements, size);


	// calculate the block size
	int blockHeightA = min(MAX_BLOCK_SIZE, Am);
	int blockWidthAHeightB = min(MAX_BLOCK_SIZE, An);
	int blockWidthB = min(MAX_BLOCK_SIZE, Bn);

	dim3 dimBlock(max(blockWidthAHeightB, blockWidthB), max(blockHeightA, blockWidthAHeightB));

	// calculate the grid size
	dim3 dimGrid((int)ceil((double)dev_B.width / dimBlock.x), (int)ceil((double)dev_A.height / dimBlock.y));

	matMulKernel <<< dimGrid, dimBlock >>> (dev_A, dev_B, dev_res, blockHeightA, blockWidthAHeightB, blockWidthB); // calling the kernel

	hipDoubleComplex* res = (hipDoubleComplex*)malloc(size);

	// copy the result to the given array
	hipMemcpy(res, dev_res.elements, size, hipMemcpyDeviceToHost);

	hipFree(dev_A.elements);
	hipFree(dev_B.elements);
	hipFree(dev_res.elements);

	return res;
}

__host__
void Matrix2::gpuMultIn(Matrix2& A, Matrix2& B, Matrix2& res) {

	// copy the elements from A to an array
	size_t size = A.m * A.n * sizeof(hipDoubleComplex);
	hipDoubleComplex* Alements = (hipDoubleComplex*)malloc(size);

	for (int i = 0; i < A.m; ++i) {
		for (int j = 0; j < A.n; ++j) {
			Alements[j + i * A.n] = complexToCudaComplex(A.entry(i, j));
		}
	}

	// copy the elements from B to an array
	size = B.m * B.n * sizeof(hipDoubleComplex);
	hipDoubleComplex* Blements = (hipDoubleComplex*)malloc(size);


	for (int i = 0; i < B.m; ++i) {
		for (int j = 0; j < B.n; ++j) {
			Blements[j + i * B.n] = complexToCudaComplex(B.entry(i, j));
		}
	}

	hipDoubleComplex* reslements = gpuMultArrs(Alements, A.m, A.n, Blements, B.n);

	free(Alements);
	free(Blements);


	// copy the result from the array to res
	for (int i = 0; i < A.m; ++i) {
		for (int j = 0; j < B.n; ++j) {
			res.entry(i, j) = cudaComplexToComplex(reslements[j + i * B.n]);
		}
	}

	free(reslements);
}

__host__
void init() {
	int* x;
	hipMalloc(&x, sizeof(int));
	hipFree(x);
}