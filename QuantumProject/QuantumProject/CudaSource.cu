#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include "CudaHeader.cuh"

__host__ 
inline hipDoubleComplex complexToCudaComplex(complex_t z) {
	return make_hipDoubleComplex(z.real(), z.imag());
}

__host__ 
inline complex_t cudaComplexToComplex(hipDoubleComplex z) {
	return complex_t(z.x, z.y);
}


__global__
void matAdd(hipDoubleComplex* A, hipDoubleComplex*B, hipDoubleComplex* res, int m, int n, int workPerThread, int threadUntil) {

	int i = min(threadIdx.x, threadUntil) * (workPerThread + 1) + max(threadIdx.x - threadUntil, 0) * workPerThread;

	int k;

	for (k = i; k < i + workPerThread; ++k) {
		res[i + k] = hipCadd(A[i + k], B[i + k]);
	}

	if (threadIdx.x < threadUntil) {
		res[i + k] = hipCadd(A[i + k], B[i + k]);
	}

}



__host__
void Matrix::gpuAddIn(Matrix& A, Matrix& B, Matrix& res) {
	hipDoubleComplex* A_vals, * B_vals, *res_vals;

	int m = A.m, n = A.n;

	int A_valsLength = m * n * sizeof(hipDoubleComplex);
	int B_valsLength = A_valsLength;
	int res_valsLength = A_valsLength;

	A_vals = (hipDoubleComplex*)malloc(A_valsLength);
	B_vals = (hipDoubleComplex*)malloc(B_valsLength);
	res_vals = (hipDoubleComplex*)malloc(res_valsLength);


	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			A_vals[i + m * j] = complexToCudaComplex(A.entry(i, j));
		}
	}

	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			B_vals[i + m * j] = complexToCudaComplex(B.entry(i, j));
		}
	}

	hipDoubleComplex* dev_A, * dev_B, * dev_res;

	hipMalloc(&dev_A, A_valsLength);
	hipMalloc(&dev_B, B_valsLength);
	hipMalloc(&dev_res, res_valsLength);

	hipMemcpy(dev_A, A_vals, A_valsLength, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B_vals , B_valsLength, hipMemcpyHostToDevice);


	int numbr_of_threads = 80;

	int workPerThread = m * n / numbr_of_threads;

	matAdd <<< n, numbr_of_threads >>> (dev_A, dev_B, dev_res, m, n, workPerThread, m * n % numbr_of_threads);

	hipMemcpy(res_vals, dev_res, res_valsLength, hipMemcpyDeviceToHost);


	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			res.entry(i, j) = cudaComplexToComplex(res_vals[i + m * j]);
		}
	}
}


__global__
void matMult(hipDoubleComplex* A, int Am, int An, hipDoubleComplex* B, int Bm, int Bn, hipDoubleComplex* res) {
	// i, k, j

}

__host__
void Matrix::gpuMultIn(Matrix& A, Matrix& B, Matrix& res) {
	hipDoubleComplex* A_vals, * B_vals, * res_vals;

	int m = A.m, n = A.n;

	int A_valsLength = m * n * sizeof(hipDoubleComplex);
	int B_valsLength = A_valsLength;
	int res_valsLength = A_valsLength;

	A_vals = (hipDoubleComplex*)malloc(A_valsLength);
	B_vals = (hipDoubleComplex*)malloc(B_valsLength);
	res_vals = (hipDoubleComplex*)malloc(res_valsLength);


	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			A_vals[i + m * j] = complexToCudaComplex(A.entry(i, j));
		}
	}

	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			B_vals[i + m * j] = complexToCudaComplex(B.entry(i, j));
		}
	}

	hipDoubleComplex* dev_A, * dev_B, * dev_res;

	hipMalloc(&dev_A, A_valsLength);
	hipMalloc(&dev_B, B_valsLength);
	hipMalloc(&dev_res, res_valsLength);

	hipMemcpy(dev_A, A_vals, A_valsLength, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B_vals, B_valsLength, hipMemcpyHostToDevice);


	int numbr_of_threads = 80;

	int workPerThread = m * n / numbr_of_threads;

	matMult << < n, numbr_of_threads >> > (dev_A, dev_B, dev_res, m, n, workPerThread, m * n % numbr_of_threads);

	hipMemcpy(res_vals, dev_res, res_valsLength, hipMemcpyDeviceToHost);


	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			res.entry(i, j) = cudaComplexToComplex(res_vals[i + m * j]);
		}
	}
}