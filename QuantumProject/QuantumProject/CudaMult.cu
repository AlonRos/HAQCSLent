#include "hip/hip_runtime.h"
#include "CudaHeader.cuh"
#include <chrono>


typedef struct {
	int width;
	int height;
	int stride;
	double* elements;

} GpuMatrix;

__device__ double GetElement(GpuMatrix& A, int row, int col) {
	return A.elements[row * A.stride + col];
}
__device__ void SetElement(GpuMatrix& A, int row, int col, double value) {
	A.elements[row * A.stride + col] = value;
}

__device__ GpuMatrix GetSubMatrix(GpuMatrix& A, int row, int col, int blockHeightA, int blockWidthA)
{
	GpuMatrix Asub;
	Asub.width = blockWidthA;
	Asub.height = blockHeightA;
	Asub.stride = A.stride;
	Asub.elements = &A.elements[A.stride * blockHeightA * row + blockWidthA * col];
	return Asub;
}

__global__ void MatMulKernel(GpuMatrix A, GpuMatrix B, GpuMatrix C, int blockHeightA, int blockWidthAHeightB, int blockWidthB) {
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	GpuMatrix subC = GetSubMatrix(C, blockRow, blockCol, blockHeightA, blockWidthB);

	double Cvalue = 0;

	int row = threadIdx.y;
	int col = threadIdx.x;

	for (int i = 0; i < A.width / blockWidthAHeightB; ++i) {
		GpuMatrix subA = GetSubMatrix(A, blockRow, i, blockHeightA, blockWidthAHeightB);

		GpuMatrix subB = GetSubMatrix(B, i, blockCol, blockWidthAHeightB, blockWidthB);

		__shared__ double As[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
		__shared__ double Bs[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];

		if (row < subA.height && col < subA.width) {
			As[row][col] = GetElement(subA, row, col);
		}

		if (row < subB.height && col < subB.width) {
			Bs[row][col] = GetElement(subB, row, col);
		}

		__syncthreads();
		
		if (row < subC.height && col < subC.width) {
			for (int j = 0; j < blockWidthAHeightB; ++j)
				Cvalue += As[row][j] * Bs[j][col];
		}

		__syncthreads();
	}

	if (row < subC.height && col < subC.width) {
		SetElement(subC, row, col, Cvalue);
	}
}

__host__
double* gpuMultDouble(double* A, int Am, int An, double* B, int Bn) {
    GpuMatrix dev_A, dev_B, dev_res;
    int Bm = An, resm = Am, resn = Bn;

    dev_A.width = dev_A.stride = An;
    dev_A.height = Am;
    size_t size = dev_A.width * dev_A.height * sizeof(double);
    hipMalloc(&dev_A.elements, size);
    hipMemcpy(dev_A.elements, A, size, hipMemcpyHostToDevice);

    dev_B.width = dev_B.stride = Bn;
    dev_B.height = Bm;
    size = dev_B.width * dev_B.height * sizeof(double);
    hipMalloc(&dev_B.elements, size);
    hipMemcpy(dev_B.elements, B, size, hipMemcpyHostToDevice);


    dev_res.width = dev_res.stride = resn;
    dev_res.height = resm;
    size = dev_res.width * dev_res.height * sizeof(double);
    hipMalloc(&dev_res.elements, size);


	int blockHeightA = min(MAX_BLOCK_SIZE, Am);
	int blockWidthAHeightB = min(MAX_BLOCK_SIZE, An);
	int blockWidthB = min(MAX_BLOCK_SIZE, Bn);

	dim3 dimBlock(max(blockWidthAHeightB, blockWidthB), max(blockHeightA, blockWidthAHeightB));

    dim3 dimGrid(ceil((double)dev_B.width / dimBlock.x), ceil((double)dev_A.height / dimBlock.y));

    MatMulKernel << <dimGrid, dimBlock >> > (dev_A, dev_B, dev_res, blockHeightA, blockWidthAHeightB, blockWidthB);

    double* res = (double*)malloc(size);

    hipMemcpy(res, dev_res.elements, size, hipMemcpyDeviceToHost);

    hipFree(dev_A.elements);
    hipFree(dev_B.elements);
    hipFree(dev_res.elements);

    return res;
}

__host__
void Matrix2::gpuMultIn(Matrix2& A, Matrix2& B, Matrix2& res) {
	size_t size = A.m * A.n * sizeof(double);

	double* realA = (double*) malloc(size);
    double* imagA = (double*) malloc(size);


	for (int i = 0; i < A.m; ++i) {
		for (int j = 0; j < A.n; ++j) {
            realA[j + i * A.n] = A.entry(i, j).real();
            imagA[j + i * A.n] = A.entry(i, j).imag();
		}
	}

    size = B.m * B.n * sizeof(double);
    double* realB = (double*)malloc(size);
    double* imagB = (double*)malloc(size);

	
    for (int i = 0; i < B.m; ++i) {
        for (int j = 0; j < B.n; ++j) {
            realB[j + i * B.n] = B.entry(i, j).real();
            imagB[j + i * B.n] = B.entry(i, j).imag();
        }
    }

    double* realArealB = gpuMultDouble(realA, A.m, A.n, realB, B.n);
    double* imagAimagB = gpuMultDouble(imagA, A.m, A.n, imagB, B.n);

    double* realAimagB = gpuMultDouble(realA, A.m, A.n, imagB, B.n);
    double* imagArealB = gpuMultDouble(imagA, A.m, A.n, realB, B.n);
    
	for (int i = 0; i < A.m; ++i) {
		for (int j = 0; j < B.n; ++j) {
            res.entry(i, j) = complex_t(realArealB[j + i * B.n] - imagAimagB[j + i * B.n], realAimagB[j + i * B.n] + imagArealB[j + i * B.n]);
		}
	}
}

__host__
void init() {
	int* x;
	hipMalloc(&x, sizeof(int));
	hipFree(x);
}