#include "hip/hip_runtime.h"
#include "CudaHeader.cuh"


__global__
void matAddKernel(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* res, int m, int n, int workPerThread, int threadUntil) {
	int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;

	int i = min(threadIndex, threadUntil) * (workPerThread + 1) + max(threadIndex - threadUntil, 0) * workPerThread;

	int k;

	for (k = i; k < i + workPerThread; ++k) {
		res[k] = hipCadd(A[k], B[k]);
	}


	if (threadIndex < threadUntil) {
		res[k] = hipCadd(A[k], B[k]);
	}

}

__host__
void Matrix2::gpuAddIn(Matrix2& A, Matrix2& B, Matrix2& res) {
	hipDoubleComplex* A_vals, * B_vals, * res_vals;

	int m = A.m, n = A.n;

	int A_valsLength = m * n * sizeof(hipDoubleComplex);
	int B_valsLength = A_valsLength;
	int res_valsLength = A_valsLength;

	A_vals = (hipDoubleComplex*)malloc(A_valsLength);
	B_vals = (hipDoubleComplex*)malloc(B_valsLength);
	res_vals = (hipDoubleComplex*)malloc(res_valsLength);
	
	
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			A_vals[j + i * n] = complexToCudaComplex(A.entry(i, j));
		}
	}
	
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			B_vals[j + i * n] = complexToCudaComplex(B.entry(i, j));
		}
	}

	hipDoubleComplex* dev_A, * dev_B, * dev_res;

	hipMalloc(&dev_A, A_valsLength);
	hipMalloc(&dev_B, B_valsLength);
	hipMalloc(&dev_res, res_valsLength);

	hipMemcpy(dev_A, A_vals, A_valsLength, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B_vals, B_valsLength, hipMemcpyHostToDevice);


	int number_of_threads = 1024;

	int number_of_blocks = 128;

	int workPerThread = (m * n) / number_of_threads;

	matAddKernel << < number_of_blocks, number_of_threads / number_of_blocks >> > (dev_A, dev_B, dev_res, m, n, workPerThread, (m * n) % number_of_threads);

	hipMemcpy(res_vals, dev_res, res_valsLength, hipMemcpyDeviceToHost);


	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			res.entry(i, j) = cudaComplexToComplex(res_vals[j + i * n]);
		}
	}

	free(A_vals);
	free(B_vals);
	free(res_vals);


	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_res);

}