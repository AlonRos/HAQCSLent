#include "hip/hip_runtime.h"
#include "CudaHeader.cuh"
#include <chrono>


typedef struct {
	int width;
	int height;
	int stride;
	hipDoubleComplex* elements;

} GpuMatrix;

__device__ hipDoubleComplex GetElement(GpuMatrix& A, int row, int col) {
	return A.elements[row * A.stride + col];
}
__device__ void SetElement(GpuMatrix& A, int row, int col, hipDoubleComplex value) {
	A.elements[row * A.stride + col] = value;
}

__device__ GpuMatrix GetSubMatrix(GpuMatrix& A, int row, int col, int blockHeightA, int blockWidthA)
{
	GpuMatrix Asub;
	Asub.width = blockWidthA;
	Asub.height = blockHeightA;
	Asub.stride = A.stride;
	Asub.elements = &A.elements[A.stride * blockHeightA * row + blockWidthA * col];
	return Asub;
}

__global__ void MatMulKernel(GpuMatrix A, GpuMatrix B, GpuMatrix C, int blockHeightA, int blockWidthAHeightB, int blockWidthB) {
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	GpuMatrix subC = GetSubMatrix(C, blockRow, blockCol, blockHeightA, blockWidthB);

	hipDoubleComplex Cvalue = make_hipDoubleComplex(0, 0);

	int row = threadIdx.y;
	int col = threadIdx.x;

	for (int i = 0; i < A.width / blockWidthAHeightB; ++i) {
		GpuMatrix subA = GetSubMatrix(A, blockRow, i, blockHeightA, blockWidthAHeightB);

		GpuMatrix subB = GetSubMatrix(B, i, blockCol, blockWidthAHeightB, blockWidthB);

		__shared__ hipDoubleComplex As[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
		__shared__ hipDoubleComplex Bs[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];

		if (row < subA.height && col < subA.width) {
			As[row][col] = GetElement(subA, row, col);
		}

		if (row < subB.height && col < subB.width) {
			Bs[row][col] = GetElement(subB, row, col);
		}

		__syncthreads();
		
		if (row < subC.height && col < subC.width) {
			for (int j = 0; j < blockWidthAHeightB; ++j)
				Cvalue = hipCadd(Cvalue, hipCmul(As[row][j], Bs[j][col]));
		}

		__syncthreads();
	}

	if (row < subC.height && col < subC.width) {
		SetElement(subC, row, col, Cvalue);
	}
}

__host__
hipDoubleComplex* gpuMultComplex(hipDoubleComplex* A, int Am, int An, hipDoubleComplex* B, int Bn) {
    GpuMatrix dev_A, dev_B, dev_res;
    int Bm = An, resm = Am, resn = Bn;

    dev_A.width = dev_A.stride = An;
    dev_A.height = Am;
    size_t size = dev_A.width * dev_A.height * sizeof(hipDoubleComplex);
    hipMalloc(&dev_A.elements, size);
    hipMemcpy(dev_A.elements, A, size, hipMemcpyHostToDevice);

    dev_B.width = dev_B.stride = Bn;
    dev_B.height = Bm;
    size = dev_B.width * dev_B.height * sizeof(hipDoubleComplex);
    hipMalloc(&dev_B.elements, size);
    hipMemcpy(dev_B.elements, B, size, hipMemcpyHostToDevice);


    dev_res.width = dev_res.stride = resn;
    dev_res.height = resm;
    size = dev_res.width * dev_res.height * sizeof(hipDoubleComplex);
    hipMalloc(&dev_res.elements, size);


	int blockHeightA = min(MAX_BLOCK_SIZE, Am);
	int blockWidthAHeightB = min(MAX_BLOCK_SIZE, An);
	int blockWidthB = min(MAX_BLOCK_SIZE, Bn);

	dim3 dimBlock(max(blockWidthAHeightB, blockWidthB), max(blockHeightA, blockWidthAHeightB));

    dim3 dimGrid(ceil((double)dev_B.width / dimBlock.x), ceil((double)dev_A.height / dimBlock.y));

    MatMulKernel << <dimGrid, dimBlock >> > (dev_A, dev_B, dev_res, blockHeightA, blockWidthAHeightB, blockWidthB);

    hipDoubleComplex* res = (hipDoubleComplex*)malloc(size);

    hipMemcpy(res, dev_res.elements, size, hipMemcpyDeviceToHost);

    hipFree(dev_A.elements);
    hipFree(dev_B.elements);
    hipFree(dev_res.elements);

    return res;
}

__host__
void Matrix2::gpuMultIn(Matrix2& A, Matrix2& B, Matrix2& res) {
	size_t size = A.m * A.n * sizeof(hipDoubleComplex);

	hipDoubleComplex* Alement = (hipDoubleComplex*) malloc(size);


	for (int i = 0; i < A.m; ++i) {
		for (int j = 0; j < A.n; ++j) {
			Alement[j + i * A.n] = complexToCudaComplex(A.entry(i, j));
            //imagA[j + i * A.n] = A.entry(i, j).imag();
		}
	}

    size = B.m * B.n * sizeof(hipDoubleComplex);
    hipDoubleComplex* Blements = (hipDoubleComplex*)malloc(size);

	
    for (int i = 0; i < B.m; ++i) {
        for (int j = 0; j < B.n; ++j) {
			Blements[j + i * B.n] = complexToCudaComplex(B.entry(i, j));
        }
    }

    hipDoubleComplex* reslement = gpuMultComplex(Alement, A.m, A.n, Blements, B.n);
    
	for (int i = 0; i < A.m; ++i) {
		for (int j = 0; j < B.n; ++j) {
            res.entry(i, j) = cudaComplexToComplex(reslement[j + i * B.n]);
		}
	}
}

__host__
void init() {
	int* x;
	hipMalloc(&x, sizeof(int));
	hipFree(x);
}